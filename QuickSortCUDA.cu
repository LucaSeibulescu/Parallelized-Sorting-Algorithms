
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

 void printArr(int arr[], int n) {
    int i;
    for(i = 0; i < n; ++i) {
        printf("%d ", arr[i]);
    }
}

__device__ int dataSize;

__global__ void partition(int *arr, int *arrLow, int *arrHigh, int n) {
    int z = blockIdx.x * blockDim.x + threadIdx.x;
    dataSize = 0;
    __syncthreads();
    if(z < n) {
        int end = arrHigh[z];
        int start = arrLow[z];
        int x = arr[end];
        int i = (start - 1);
        int temp;
        for(int j = start; j <= end - 1; j++) {
            if(arr[j] <= x) {
                i++;
                temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
            }
        }
        temp = arr[i + 1];
        arr[i + 1] = arr[end];
        arr[end] = temp;
        int k = (i + 1);
        if(k - 1 > start) {
            int index = atomicAdd(&dataSize, 1);
            arrLow[index] = start;
            arrHigh[index] = k - 1;  
        }
        if(k + 1 < end) {
            int index = atomicAdd(&dataSize, 1);
            arrLow[index] = k+1;
            arrHigh[index] = end; 
        }
    }
}
 
void quickSortIterative(int arr[], int start, int end, int threads, int numelements) {
    clock_t startTime, endTime;
    int lowStack[end - start + 1], highStack[end - start + 1];
 
    int top = -1, *data, *data_low, *data_high;
 
    lowStack[++top] = start;
    highStack[top] = end;

    startTime = clock();
    hipMalloc(&data, (end - start + 1)*sizeof(int));
    hipMemcpy(data, arr, (end - start + 1)*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&data_low, (end - start + 1)*sizeof(int));
    hipMemcpy(data_low, lowStack, (end - start + 1)*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&data_high, (end - start + 1)*sizeof(int));
    hipMemcpy(data_high, highStack, (end - start + 1)*sizeof(int), hipMemcpyHostToDevice);
    endTime = clock();
    double communication = ((double) (endTime - startTime)) / CLOCKS_PER_SEC;
    int blocks = numelements/threads;
    int n_i = 1; 
    startTime = clock();
    while(n_i > 0) {
        partition<<<blocks,threads>>>( data, data_low, data_high, n_i);
        int answer;
        hipMemcpyFromSymbol(&answer, HIP_SYMBOL(dataSize), sizeof(int), 0, hipMemcpyDeviceToHost); 
        if(answer < 1024) {
            threads = answer;
        }
        else {
            threads = 1024;
            blocks = answer/threads + (answer%threads==0?0:1);
        }
        n_i = answer;
        hipMemcpy(arr, data, (end - start + 1)*sizeof(int), hipMemcpyDeviceToHost);
    }
    endTime = clock();
    double computation = ((double) (endTime - startTime)) / CLOCKS_PER_SEC;
    //printf("Communication Time: %f, Computation Time: %f", communication, computation);
    printf("%f,%f,", computation, communication);
}
 

 
int main(int argc, char* argv[]) {
    int numelements;
    int threads;
    char array_type[8];
    clock_t startTotal, endTotal;
    numelements = atoi(argv[1]);
    startTotal = clock();
    int arr[numelements];
    if(strcmp(argv[2],"reverse") == 0) {
        strcpy(array_type, "reverse");
        for(int i = numelements - 1; i >= 0; i--) {
            arr[i] = i;
        }
    }
    else if(strcmp(argv[2],"sorted") == 0) {
        strcpy(array_type, "sorted");
        for(int i = 0; i < numelements; i++) {
            arr[i] = i;
        }
    }
    else if(strcmp(argv[2],"random") == 0) {
        strcpy(array_type, "random");
        srand(time(NULL));
        for(int i = 0; i < numelements; i++) {
            arr[i] = rand() % numelements;
        }
    }
    threads = atoi(argv[3]);
    int n = sizeof(arr) / sizeof(*arr);
    quickSortIterative(arr, 0, n - 1, threads, numelements);
    endTotal = clock();
    double total_time = ((double) (endTotal - startTotal)) / CLOCKS_PER_SEC;
    //printArr(arr, n);
    //printf("\nTotal Time: %f", total_time);
    printf("%d,%d,%s\n", threads, numelements, array_type);
    return 0;
}